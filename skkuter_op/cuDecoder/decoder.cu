#include "hip/hip_runtime.h"
#include <torch/types.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <torch/extension.h>

#include <torch/torch.h>
#include "ATen/ATen.h"
#include <cmath>

#define ELEMENT_TYPE torch::kBFloat16


#define CHECK_CUDA_ERROR(val) check((val), #val, __FILE__, __LINE__)
void check(hipError_t err, const char* const func, const char* const file,
           const int line)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << " " << func << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}

#define CHECK_LAST_CUDA_ERROR() checkLast(__FILE__, __LINE__)
void checkLast(const char* const file, const int line)
{
    hipError_t const err{hipGetLastError()};
    if (err != hipSuccess)
    {
        std::cerr << "CUDA Runtime Error at: " << file << ":" << line
                  << std::endl;
        std::cerr << hipGetErrorString(err) << std::endl;
        // We don't exit when we encounter CUDA errors in this example.
        // std::exit(EXIT_FAILURE);
    }
}



/*
    KHAN

    This kernel is responsible for carrying out the attention between 1 query and all the keys
    The kernel will first multiply Q and transposed_k 
    Then it will scale the dot product
    Then it will apply the softmax
    Then it will multiply the softmax with the values
    Finally it will store the result in the output tensor

*/


#define CONVERT(x) __float2bfloat16(x)
#define BTYPE __hip_bfloat16

__global__
void attention_forward_kernel(BTYPE* Q, BTYPE* K, BTYPE* V, BTYPE* O, BTYPE* mask,
                                float div,
                                int Tc, int Bc,
                                int d, int qN, int kN){

    int batch = gridDim.z;
    int nH = gridDim.y;
    int tx = threadIdx.x;

    int current_batch = blockIdx.z;
    int current_head = blockIdx.y;
    int current_seq = blockIdx.x;

    
    int q_offset = (current_batch * nH * qN * d) + (current_head * qN * d) + (current_seq * d);  
    int o_offset = (current_batch * nH * qN * d) + (current_head * qN * d) + (current_seq * d);  
    int k_offset = (current_batch * nH * kN * d) + (current_head * kN * d);
    int v_offset = (current_batch * nH * kN * d) + (current_head * kN * d);
    int mask_offset = (current_batch * kN * qN) + (current_seq * kN);

    BTYPE* q = Q + q_offset; // (1 x d)
    BTYPE* o = O + o_offset; // (1 x d)
    BTYPE* v = V + v_offset; // (kN x d)
    BTYPE* k = K + k_offset; // (d x kN)

    BTYPE divv = CONVERT(div);
    extern __shared__ BTYPE sram[];
    BTYPE* shared_o = sram;

    
    for(int i = 0; i < Tc; i++){
        BTYPE* k_ptr = k + (i * Bc); // (d x Bc)
        BTYPE sum = CONVERT(0.0f);

        //skip if the index is out of bound
        if(i * Bc + tx >= kN)
            continue;

        for(int j = 0; j < d; j++){
            sum += q[j] * k_ptr[j * kN + tx];
        }

        shared_o[i*Bc+tx] = sum * divv;
        shared_o[i*Bc+tx] += mask[mask_offset + i*Bc+tx];
        
    }

    __syncthreads();

    //Simply compute the softmax
    BTYPE sum = CONVERT(0.0f);
    BTYPE max = -CUDART_INF_BF16;
    for(int i = 0; i < kN; i++){;
        if(shared_o[i] > max){
            max = shared_o[i];
        }
    }

    for(int i = 0; i < kN; i++){
        shared_o[i] = hexp(shared_o[i] - max);
        sum += shared_o[i];        
    }


    //I have bc number of threads and the row size is kN
    int elements_per_thread = (kN + Bc - 1) / Bc;
    int my_index = tx * elements_per_thread;

    for(int i = 0; i < elements_per_thread; i++){
        shared_o[my_index + i] /= sum;
    }

    __syncthreads();

    if(tx == 0){
        for(int i = 0; i < d; i++){
            BTYPE sum = CONVERT(0.0f);
            for(int j = 0; j < kN; j++){
                sum += shared_o[j] * v[j * d + i];
            }

            o[i] = sum;
        }

    }

}

int get_sram_size(){
    
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    return prop.sharedMemPerMultiprocessor;

}

torch::Tensor attention_forward(torch::Tensor Q, torch::Tensor K, torch::Tensor V, torch::Tensor mask){

    K = K.transpose(2,3);

    int M = get_sram_size();
    int batch = Q.size(0); int nH = Q.size(1); 
    int qN = Q.size(2); int kN = K.size(3); //Because k is transposed
    int d = Q.size(3);

    //For scaling the dot product
    auto div = 1.0f/std::sqrt(static_cast<float>(d));


    /* KHAN
        The number of columns in K will determine the number of threads
        But it should be multiple of 32
        Get bc such that kN is a multiple of bc
    */

    int Bc = 32; 
    int Tc = (kN + Bc - 1) / Bc;
    
    dim3 block(Bc, 1, 1);
    dim3 grid(qN,nH, batch);

    /* KHAN
        Create an O tensor of size batch x nH x qN x d
        Create the pointers for the Q, K, V and O tensors
    */

    torch::Tensor O = torch::zeros({batch, nH, qN, d}, Q.options());

    __hip_bfloat16* Q_ptr = reinterpret_cast<__hip_bfloat16*>(Q.data_ptr());
    __hip_bfloat16* K_ptr = reinterpret_cast<__hip_bfloat16*>(K.data_ptr());
    __hip_bfloat16* V_ptr = reinterpret_cast<__hip_bfloat16*>(V.data_ptr());
    __hip_bfloat16* O_ptr = reinterpret_cast<__hip_bfloat16*>(O.data_ptr());
    __hip_bfloat16* mask_ptr = reinterpret_cast<__hip_bfloat16*>(mask.data_ptr());


    /* KHAN
        Launch the kernel for attention
    */
    attention_forward_kernel<<<grid, block>>>(Q_ptr, K_ptr, V_ptr, O_ptr, mask_ptr,
        div,
        Tc, Bc,
        d, qN, kN);

    hipDeviceSynchronize();
    CHECK_LAST_CUDA_ERROR();

    return O;

    


}


void myTest(){
    torch::Tensor Q = torch::randn({1, 40, 1, 128}, ELEMENT_TYPE).to(torch::kCUDA);
    torch::Tensor Q2 = torch::empty({1, 40, 1, 128}, ELEMENT_TYPE).to(torch::kCUDA);

    torch::Tensor K = torch::randn({1, 40, 170, 128}, ELEMENT_TYPE).to(torch::kCUDA);
    torch::Tensor K2 = torch::empty({1, 40, 170, 128}, ELEMENT_TYPE).to(torch::kCUDA);
    
    torch::Tensor V = torch::randn({1, 40, 170, 128}, ELEMENT_TYPE).to(torch::kCUDA);
    torch::Tensor V2 = torch::empty({1, 40, 170, 128}, ELEMENT_TYPE).to(torch::kCUDA);

    torch::Tensor mask = torch::randn({1, 1, 1, 170}, ELEMENT_TYPE).to(torch::kCUDA);
    mask.fill_(0);
    auto O = attention_forward(Q, K, V, mask);

    Q2.copy_(Q);
    K2.copy_(K);
    V2.copy_(V);



    // //do the attention by pytorch
    auto div = std::sqrt(128);
    auto query_states = torch::matmul(Q2, K2.transpose(2,3)) / div; 
    query_states = query_states + mask;
    query_states = torch::nn::functional::softmax(query_states, torch::nn::functional::SoftmaxFuncOptions(-1.f)).to(ELEMENT_TYPE);
    auto value_states = torch::matmul(query_states, V2);

    //get the different between O and value_states
    auto diff = torch::abs(value_states - O);

    // //print the max diff
    std::cout << "Max diff: " << diff.max().item<float>() << std::endl;

}
